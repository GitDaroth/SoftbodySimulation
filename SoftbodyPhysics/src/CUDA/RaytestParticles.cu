#include "hip/hip_runtime.h"
#include "CUDA/RaytestParticles.cuh"

void raytestParticlesWithCuda(std::shared_ptr<ParticleData> particleData, float3 rayOrigin, float3 rayDircetion)
{
	raytestParticlesKernel<<<(particleData->particleCount + 511) / 512, 512>>>(*particleData, rayOrigin, rayDircetion);
}

__global__ void raytestParticlesKernel(ParticleData particleData, float3 rayOrigin, float3 rayDircetion)
{
	int particleIndex = blockIdx.x * blockDim.x + threadIdx.x;

	if (particleIndex >= particleData.particleCount)
		return;

	float3 distanceVec = rayOrigin - particleData.predictedPosition[particleIndex];
	float b = dot(rayDircetion, distanceVec);
	float c = dot(distanceVec, distanceVec) - particleData.radius * particleData.radius;

	if (b * b - c < 0)
		particleData.raytestResult[particleIndex] = -1.0f;
	else
		particleData.raytestResult[particleIndex] = -b + sqrtf(b * b - c);
}